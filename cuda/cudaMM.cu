#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <vector>
#include <iostream>
#include <string>
#include <algorithm>
#include <atomic>
#include <mutex>
#include <thread>
#include <omp.h>
#include <cstdlib>
#include <numeric>
#include <cmath>
#include <ctime> 

#define CEIL(x, y) (((x) + (y) - 1) / (y))
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define ABS(x) (((x) < (0)) ? (-x) : (x))

#define BLOCK_SIZE 4
#define MAX_TPB 32

#define MM
#define PSIZE 2000

#define EPSILON 1e-4

// #define SCHED_ROUNDROBIN
// #define SCHED_DYNAMIC
// #define SCHED_DYNAMIC2
// #define SCHED_RANDOM
// #define SCHED_ADAPTIVE
// #define SCHED_ADAPTIVE2

// using data_type = float;

std::mutex mtx;
// Define the global variable
// __device__ int d_counter = 0;

// #define USEOPENMP
// #define PRE_TRANSFER

__host__ inline hipError_t checkCuda(hipError_t status)
{
    if (status != hipSuccess)
    {
        std::cout << "CUDA Runtime Error: " << hipGetErrorString(status)
                  << std::endl;
        std::exit(EXIT_FAILURE);
    }
    return status;
}

__host__ inline unsigned gpu_scheduler_static_rr(int taskID, int ngpus)
{
    return taskID % ngpus;
}

__host__ inline unsigned gpu_scheduler_dynamic_ad(unsigned long *gpuLoad, int ngpus, int taskWeight)
{
    short looking = 1;
    unsigned chosen;
    while (looking)
    {
        unsigned occ_i;
        unsigned long load;
        unsigned long min_load = ULLONG_MAX;
        for (unsigned i = 0; i < ngpus; i++)
        {
#pragma omp atomic read
            load = gpuLoad[i];
            if (load < min_load)
            {
                min_load = load;
                occ_i = i;
            }
        }
        chosen = occ_i;
#pragma omp atomic
        gpuLoad[chosen] += taskWeight;
        looking = 0;
        break;
    }
    return chosen;
}

// This version avoids all CPU threads finding the same GPU greedily (and therefore overloading that GPU)
__host__ inline unsigned gpu_scheduler_dynamic_ad2(unsigned long *gpuLoad, int ngpus, int taskWeight)
{
    short looking = 1;
    unsigned chosen;
    while (looking)
    {
        unsigned long load;
        unsigned long min_load = ULLONG_MAX;

#pragma omp critical
        {
            for (unsigned i = 0; i < ngpus; i++)
            {
                load = gpuLoad[i];
                if (load < min_load)
                {
                    min_load = load;
                    chosen = i;
                }
            }
            gpuLoad[chosen] += taskWeight;
        }
        looking = 0;
        break;
    }
    return chosen;
}

__host__ inline unsigned gpu_scheduler_dynamic_random(unsigned *occupancies, int ngpus)
{
    const unsigned chosen = rand() % ngpus;
#pragma omp atomic
    occupancies[chosen]++;
    return chosen;
}

__host__ inline unsigned gpu_scheduler_dynamic_occ2(unsigned *occupancies, int ngpus)
{
    int chosen = -1;
    while (chosen == -1)
    {
        for (unsigned i = 0; i < ngpus; i++)
        {
#pragma omp critical
            {
                if (occupancies[i] == 0)
                {
                    occupancies[i]++;
                    chosen = i;
                }
            }
            if (chosen > -1)
                break;
        }
    }
    return chosen;
}

__host__ inline unsigned gpu_scheduler_dynamic_occ(unsigned *occupancies, int ngpus)
{
    short looking = 1;
    unsigned chosen;
    while (looking)
    {
        for (unsigned i = 0; i < ngpus; i++)
        {
            // But really, this should be a single atomic compare-and-swap
            unsigned occ_i;
#pragma omp atomic read
            occ_i = occupancies[i];
            if (occ_i == 0)
            {
                chosen = i;
#pragma omp atomic
                occupancies[chosen]++;
                looking = 0;
                break;
            }
        }
    }
    return chosen;
}

void transposeMatrix(float* matrix, int m, int n) {
    for (int i = 0; i < m; ++i) {
        for (int j = i + 1; j < n; ++j) {
            std::swap(matrix[i * n + j], matrix[j * n + i]);
        }
    }
}

// Kernel for matrix-matrix multiplication
__global__ void multiply_kernel(float *A, float *B, float *C, int rowStart, int M, int N, int K)
{
    // int i = blockIdx.y * blockDim.y + threadIdx.y;
    // int j = blockIdx.x * blockDim.x + threadIdx.x;
    // // i += rowStart;
    // printf("i inc = %d j inc = %d\n",blockDim.y*gridDim.y,blockDim.x*gridDim.x);
    for(int i = blockIdx.y * blockDim.y + threadIdx.y;i<M;i+=blockDim.y*gridDim.y){
        for(int j = blockIdx.x * blockDim.x + threadIdx.x;j<N;j+=blockDim.x*gridDim.x)
    // if(i < M and j < N)
            {
                // atomicAdd(&d_counter, 1);
                float sum = 0.0;
                
                #if defined(VECTORIZE)
                auto a = reinterpret_cast<float4*>(&A[i * K]);
                auto b = reinterpret_cast<float4*>(&B[j * K]);
                // printf("check %d %d (%p %p) (%p %p) %d\n",i*K*4,j*K*4,a,&A[i * K],b,&B[j * K],K/4);
                for (int k = 0; k < K/4; k++)
                {
                    // printf("before\n");
                    // auto a = a_4[k], b = b_4[k];
                    // printf("%f,%f,%f,%f %f,%f,%f,%f\n",a.w,a.x,a.y,a.z,b.w,b.x,b.y,b.z);
                    sum += a->x*b->x + a->y*b->y + a->z*b->z + a->w*b->w;
                    // printf("(%f,%f)\n",a->w,b->w);
                    a++;
                    b++;
                }
                
                #else
                for (int k = 0; k < K; ++k)
                    sum += A[i * K + k] * B[j * K + k];
                #endif
                C[i * N + j] = sum;

                
            // printf("\n-------------\n");
            }
    }
}

__global__ void gpu_check(int i, int d){
    printf("task %d on GPU %d done\n", i, d);
}

void printMatrix(float *mat, int m, int n){
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%0.2lf ",mat[i*n+j]);
        }
        printf("\n");
    }
    printf("\n");
}

__global__ void printMatrixKernel(float* matrix, int width, int height) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx < width && idy < height) {
        printf("Element at [%d, %d]: %f\n", idy, idx, matrix[idy * width + idx]);
    }
}

void printMatrixGPU(float *mat, int m, int n){
    dim3 blocksPerGrid((n+min(MAX_TPB,n)-1)/min(MAX_TPB,n), (m+min(MAX_TPB,m)-1)/min(MAX_TPB,m));
    dim3 threadsPerBlock(min(MAX_TPB,n), min(MAX_TPB,m)); // Assuming width and height are within max threads per block limit 
    printMatrixKernel<<<blocksPerGrid, threadsPerBlock>>>(mat, n, m);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
}

auto clk = std::chrono::high_resolution_clock::now();

void start_timer(){
    clk = std::chrono::high_resolution_clock::now();
}

void end_timer(std::string func){
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - clk);
    std::cout<<func<<" took "<<1.0e-9 * duration.count()<<" seconds\n";
}

void joinThreads(std::vector<std::thread> &threads){
    #if not defined(USEOPENMP)
    for (auto &thread: threads){
        thread.join();
    }
    threads.clear();
    #endif
}

// Function to calculate mean of chunk sizes
double calculateMean(const std::vector<int>& chunkSizes) {
    double sum = 0.0;
    for (int size : chunkSizes) {
        sum += size;
    }
    std::cout << "Mean: " << sum / chunkSizes.size() << "\t";
    return sum / chunkSizes.size();
}

// Function to calculate standard deviation of chunk sizes
double calculateStandardDeviation(const std::vector<int>& chunkSizes, double mean) {
    double sum = 0.0;
    for (int size : chunkSizes) {
        sum += std::pow(size - mean, 2);
    }
    std::cout << "Standard deviation: " << std::sqrt(sum / chunkSizes.size()) << std::endl;
    return std::sqrt(sum / chunkSizes.size());
}

// Function to calculate chunk sizes from start indices
std::vector<int> calculateChunkSizes(const std::vector<int>& startIndexes, int n) {
    std::vector<int> chunkSizes;
    for (size_t i = 0; i < startIndexes.size(); ++i) {
        if (i == startIndexes.size() - 1) {
            chunkSizes.push_back(n - startIndexes[i]);  // Last chunk goes to the end of the array
        } else {
            chunkSizes.push_back(startIndexes[i + 1] - startIndexes[i]);
        }
    }
    return chunkSizes;
}

std::vector<int> generateUniformChunkStartIndices(int n, int m) {
    std::vector<int> chunkSizes(m, 1); // Start each chunk with at least one element
    int remainingElements = n - m;    // Elements left after giving 1 to each chunk

    srand(101);
    // Distribute the remaining elements randomly
    for (int i = 0; i < remainingElements; ++i) {
        int chunkIndex = rand() % m;
        chunkSizes[chunkIndex]++;
    }

    // Calculate the starting indices
    std::vector<int> startIndexes(m);
    std::partial_sum(chunkSizes.begin(), chunkSizes.end() - 1, startIndexes.begin() + 1);

    return startIndexes;
}

std::vector<int> generateEqualChunkStartIndices(int n, int m) {
    std::vector<int> startIndexes;
    int baseSize = n / m;               // Base size of each chunk
    int remainder = n % m;              // Remainder to be distributed
    int startIndex = 0;

    // Generate starting indices based on uniform chunk sizes
    for (int i = 0; i < m; ++i) {
        startIndexes.push_back(startIndex);
        int currentChunkSize = baseSize + (i < remainder ? 1 : 0);  // Distribute remainder among the first few chunks
        startIndex += currentChunkSize;
    }

    return startIndexes;
}


std::vector<int> generateRandomChunkStartIndices(int n, int m) {
    std::vector<int> chunkSizes;
    std::vector<int> startIndexes;
    int totalSize = 0;

    // Seed the random number generator
    srand(101);

    // Generate random chunk sizes
    for (int i = 0; i < m; ++i) {
        if (i == m - 1) {
            chunkSizes.push_back(n - totalSize); // Last chunk takes the remaining elements
        } else {
            int remaining = n - totalSize - (m - i - 1); // Ensure space for at least 1 element per remaining chunk
            int chunkSize = 1 + rand() % remaining;
            chunkSizes.push_back(chunkSize);
            totalSize += chunkSize;
        }
    }

    // Calculate starting indices from chunk sizes
    int startIndex = 0;
    for (int size : chunkSizes) {
        startIndexes.push_back(startIndex);
        startIndex += size;
    }

    return startIndexes;
}

int main(int argc, char **argv)
{
    int ndevs = 0;
    hipError_t error_id = hipGetDeviceCount(&ndevs);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        return 1;
    }

    // Output the number of GPUs
    printf("Number of GPUs available: %d\n", ndevs);
    int *devices = (int *)calloc(ndevs, sizeof(*devices));
    // double start_iterations, end_iterations;
    unsigned *lastGPU = NULL;

    //  int chosen[N];
    unsigned *occupancies = (unsigned *)calloc(ndevs, sizeof(*occupancies));
    unsigned long *gpuLoad = (unsigned long *)calloc(ndevs, sizeof(*gpuLoad));
    
    int timestep = 1;
    // int probSize = MAXWORK;
    int numThreads = 64;
    int numThreadsPerBlock = 256;
    // numThreads = omp_get_num_threads();
    int M = PSIZE, N = PSIZE, K = PSIZE;
    int check_result = 0;

    // srand((unsigned)time(NULL));
    float granularity = 0.9;
    if (argc <= 1)
    {
        printf("Usage bench_works [m] [n] [k] [granularity]\n");
        printf("Using default parameters\n");
    }
    else
    {
        if (argc > 1)
            M = atoi(argv[1]);
        if (argc > 2)
            N = atoi(argv[2]);
        if (argc > 3)
            K = atoi(argv[3]);
        if (argc > 4)
        {
            granularity = atof(argv[4]);
            if (granularity < 0.0 || granularity > 1.0)
            {
                fprintf(stderr, "Error: granularity must be between 0.0 and 1.0. Received %lf.\n", granularity);
                exit(1); // Exit with error code 1
            }
        }
        if (argc > 5)
            numThreadsPerBlock = atoi(argv[5]);
        if (argc > 6)
            numThreads = atoi(argv[6]);
        if (argc > 7)
            check_result = 1;
    }
    int a_size = M * K, b_size = K * N, c_size = M * N;

    int rowsPerTask = MAX(1, (1.0 - granularity) * M);
    int numTasks = CEIL(M,rowsPerTask);
    // int streams_per_gpu = CEIL(numTasks,ndevs);
    int streams_per_gpu = 32;
    numThreadsPerBlock = CEIL(1024,streams_per_gpu);
    printf("bench_works [m=%d] [n=%d] [k=%d] [numTasks=%d] [granularity=%0.2lf] [rowsPerTask=%d] [numThreads=%d] [numThreadsPerBlock=%d] [resMatSize=%0.2e] [streams_per_gpu=%d]\n",
            M, N, K, numTasks, granularity, rowsPerTask, numThreads, numThreadsPerBlock, 1.0f*c_size, streams_per_gpu);

    #if defined(SCHED_ROUNDROBIN)
    printf("gpu_scheduler_static_rr,\t");
    #elif defined(SCHED_ADAPTIVE)
    printf("gpu_scheduler_dynamic_ad,\t");
    #elif defined(SCHED_ADAPTIVE2)
    printf("gpu_scheduler_dynamic_ad2,\t");
    #elif defined(SCHED_RANDOM)
    printf("gpu_scheduler_dynamic_random,\t");
    #elif defined(SCHED_DYNAMIC)
    printf("gpu_scheduler_dynamic_occ,\t");
    #elif defined(SCHED_DYNAMIC2)
    printf("gpu_scheduler_dynamic_occ2,\t");
    #else
    printf("none 0\n");
    #endif

    #if defined(ASYN)
    printf("asyn nowait\n");
    #else
    printf("syn with wait\n");
    #endif

    float *a,*b,*c;

    checkCuda(hipHostMalloc(&a,a_size*sizeof(float)));
    checkCuda(hipHostMalloc(&b,b_size*sizeof(float)));
    checkCuda(hipHostMalloc(&c,c_size*sizeof(float)));

    int *taskWork = (int *)malloc(sizeof(int) * numTasks);

    int *chosen = (int *)malloc(sizeof(int) * numTasks);
    int *success = (int *)malloc(sizeof(int) * numTasks);

    // initialize

    for (int i = 0; i < a_size; i++)
        // a[i] = (float)rand() / RAND_MAX * 2.0 - 1.0;
        a[i] = i%4;

    for (int i = 0; i < b_size; i++)
        // b[i] = (float)rand() / RAND_MAX * 2.0 - 1.0;
        b[i] = 4+i%3;

    for (int i = 0; i < c_size; i++)
        c[i] = 0.0;

    // printMatrix(a,M,K);printMatrix(b,K,N);printMatrix(c,M,N);
    std::vector<std::vector<hipStream_t>> streams(ndevs,std::vector<hipStream_t>(streams_per_gpu));
    for(int d=0;d<ndevs;d++){
        hipSetDevice(d);
        for(int s=0;s<streams_per_gpu;s++)
            hipStreamCreate(&streams[d][s]);
    }
    std::vector<int> strm_ctr(ndevs,0);

    auto nxt_strm = [&](int& x) -> int {
        int temp;
    #if defined(USEOPENMP)
        #pragma omp critical
        {
    #endif
    #if not defined(USEOPENMP)
            std::lock_guard<std::mutex> lock(mtx);
    #endif
            temp = x;
            x = (x+1)%streams_per_gpu;
    #if defined(USEOPENMP)
        }
    #endif
        return temp;
    };

    std::vector<int> startIndexes = generateEqualChunkStartIndices(M, numTasks);;
    
    // startIndexes = generateUniformChunkStartIndices(M, numTasks);
    startIndexes = generateRandomChunkStartIndices(M, numTasks);

    // std::cout << "Starting indices of chunks: ";
    // for (int index : startIndexes) {
    //     std::cout << index << " ";
    // }
    // std::cout << std::endl;
    
    // Calculate chunk sizes from start indices
    std::vector<int> chunkSizes = calculateChunkSizes(startIndexes, M);

    calculateStandardDeviation(chunkSizes, calculateMean(chunkSizes));

    std::vector<float*> d_b_global(ndevs);

    std::vector<std::thread> threads;

    transposeMatrix(b,K,N);
    
    #if defined(VECTORIZE)
    printf("vectorized,\t");
    #else
    printf("non-vectorized,\t");
    #endif

    #if defined(USEOPENMP)
    printf("openMP,\t");
    #else
    printf("non-openMP,\t");
    #endif


    start_timer();

    #if defined(PRE_TRANSFER)
    printf("PRE TRANSFER\n");
    #if defined(USEOPENMP)
    #pragma omp parallel for schedule(static,1)
    #endif
    for(int d=0;d<ndevs;d++){
        #if not defined(USEOPENMP)
        threads.push_back(std::thread([&, d]() {
        #endif
            hipSetDevice(d);

            int nxt = nxt_strm(strm_ctr[d]);
            auto stream = streams[d][nxt];
            
            hipMallocAsync(&d_b_global[d],b_size*sizeof(float),stream);
            hipMemcpyAsync(d_b_global[d],b,b_size*sizeof(float),hipMemcpyHostToDevice,stream);
            
            hipDeviceSynchronize();
        
        #if not defined(USEOPENMP)
        }));
        #endif
    }

    joinThreads(threads);
    #else
    printf("No pre transfer\n");
    #endif
    int nextTask = ndevs;

    #if defined(USEOPENMP)
    #pragma omp parallel for schedule(static,1)
    #endif
    for (int i = 0; i < numTasks; i++){
        // printf("thread %d\ti %d\n",omp_get_thread_num(),i);
        #if not defined(USEOPENMP)
        threads.push_back(std::thread([&,i](){
        #endif
            // int start = i*rowsPerTask, end = MIN((i+1)*rowsPerTask,M);
            int start = startIndexes[i], end = (i==numTasks-1 ? M : startIndexes[i+1]);
            int nRows = end-start;
            float *d_a, *d_b, *d_c;
            int a_start, b_start, c_start, a_items, b_items, c_items, m, n, k;
            
            m=nRows; n=N; k=K;
            a_start = start*K; b_start = 0;   c_start = start*N;
            a_items = nRows*K; b_items = K*N; c_items = nRows*N;
            
            const int NNsq = c_items;

            #if defined(SCHED_ROUNDROBIN)
            const int dev = gpu_scheduler_static_rr(i, ndevs);
            #elif defined(SCHED_ADAPTIVE)
            const int dev = gpu_scheduler_dynamic_ad(gpuLoad, ndevs, NNsq);
            #elif defined(SCHED_ADAPTIVE2)
            const int dev = gpu_scheduler_dynamic_ad2(gpuLoad, ndevs, NNsq);
            #elif defined(SCHED_RANDOM)
            const int dev = gpu_scheduler_dynamic_random(occupancies, ndevs);
            #elif defined(SCHED_DYNAMIC)
            const int dev = gpu_scheduler_dynamic_occ(occupancies, ndevs);
            #elif defined(SCHED_DYNAMIC2)
            const int dev = gpu_scheduler_dynamic_occ2(occupancies, ndevs);
            #else
            const int dev = 0;
            #endif
            if (dev != -1)
                chosen[i] = dev;
            success[i] = 0;

            int d = chosen[i]; // assert(0 <= chosen[i] <= ndevs-1)

            devices[d]++;

            int nxt = nxt_strm(strm_ctr[d]);
            // printf("dev %d [%d] (%d,%d) GPU, stream: [%d, %d]\n",d,i,start,end,d,nxt);
            
            hipSetDevice(d);
            auto stream = streams[d][nxt];

            hipMallocAsync(&d_a,a_items*sizeof(float),stream);
            hipMemcpyAsync(d_a,a+a_start,a_items*sizeof(float),hipMemcpyHostToDevice,stream);
            
            #if not defined(PRE_TRANSFER)
            hipMallocAsync(&d_b,b_items*sizeof(float),stream);
            hipMemcpyAsync(d_b,b+b_start,b_items*sizeof(float),hipMemcpyHostToDevice,stream);
            #endif
            
            hipMallocAsync(&d_c,c_items*sizeof(float),stream);
            hipMemcpyAsync(d_c,c+c_start,c_items*sizeof(float),hipMemcpyHostToDevice,stream);
            
            
            int blk_x = min(MAX_TPB,n), blk_y = min(MAX_TPB,m);
            // dim3 blocksPerGrid(CEIL(n,blk_x), CEIL(m,blk_y));
            // dim3 threadsPerBlock(blk_x, blk_y); // Assuming width and height are within max threads per block limit 
            dim3 blocksPerGrid(CEIL(m*n,numThreadsPerBlock),1);
            dim3 threadsPerBlock(numThreadsPerBlock,1);
            // dim3 blocksPerGrid(1,1);
            // dim3 threadsPerBlock(1,1);
            // printf(" %d %d\n",blocksPerGrid.x,threadsPerBlock.x);
            
            #if defined(PRE_TRANSFER)
            multiply_kernel<<<blocksPerGrid,threadsPerBlock,0,stream>>>(d_a,d_b_global[d],d_c,start,m,n,k);
            #else
            multiply_kernel<<<blocksPerGrid,threadsPerBlock,0,stream>>>(d_a,d_b,d_c,start,m,n,k);
            #endif
            hipMemcpyAsync(c+c_start,d_c,c_items*sizeof(float),hipMemcpyDeviceToHost,stream);

            hipFreeAsync(d_a,stream);
            #if not defined(PRE_TRANSFER)
            hipFreeAsync(d_b,stream);
            #endif
            hipFreeAsync(d_c,stream);
            
            // gpu_check<<<1,1,0,stream>>>(i,d);

            #if defined(SCHED_RANDOM) || defined(SCHED_DYNAMIC) || defined(SCHED_DYNAMIC2)
            success[i] = 1;
            hipStreamSynchronize(stream);
            occupancies[d]--;
            #endif
            #if defined(SCHED_ADAPTIVE) || defined(SCHED_ADAPTIVE2)
            hipStreamSynchronize(stream);
            success[i] = 1;
            gpuLoad[d] -= NNsq;
            // nextTask assignedTo the GPU just freed                                                                                                                                                                      
            int myTask;
            #pragma omp atomic capture 
            myTask = nextTask++;
            if(myTask < numTasks) chosen[myTask] = d;
            #endif
            // printf("dev %d [%d] (%d,%d) GPU, stream: [%d, %d]\n",d,i,start,end,d,nxt);
        
        #if not defined(USEOPENMP)
        }));
        #endif
    }
    
    joinThreads(threads);

    #if defined(USEOPENMP)
    #pragma omp parallel for schedule(static,1)
    #endif
    for(int d=0;d<ndevs;d++){
        #if not defined(USEOPENMP)
        threads.push_back(std::thread([&, d]()
        {
        #endif
            hipSetDevice (d);
            hipDeviceSynchronize();
        #if not defined(USEOPENMP)
        }));
        #endif
    }

    joinThreads(threads);

    #if defined(PRE_TRANSFER)
    #if defined(USEOPENMP)
    #pragma omp parallel for schedule(static,1)
    #endif
    for(int d=0;d<ndevs;d++){
        #if not defined(USEOPENMP)
        threads.push_back(std::thread([&, d]() {
        #endif
            hipSetDevice (d);
            int nxt = nxt_strm(strm_ctr[d]);
            auto stream = streams[d][nxt];
            hipFreeAsync(d_b_global[d],stream);
            hipDeviceSynchronize();
        #if not defined(USEOPENMP)
        }));
        #endif
    }
    #endif
    
    joinThreads(threads);

    end_timer("GPU multiplication");
    transposeMatrix(b,K,N);

    std::vector<int> percent(ndevs,0);
    for(int i=0;i<numTasks;i++) percent[chosen[i]]++;
    for(int i=0;i<ndevs;i++) printf("GPU %d: %0.2lf  ",i,(double)percent[i]/numTasks);
    printf("\n"); 

    if(check_result){
        float *c_cpu;
        // checkCuda(hipHostMalloc(&c_cpu,c_size*sizeof(float)));
        c_cpu = (float*)malloc(c_size*sizeof(float));
        start_timer();
        printf("GPU Done... now checking correctness\n");
        for (int ii = 0; ii < M; ii++)
            for (int jj = 0; jj < N; jj++){
                c_cpu[ii * N + jj] = 0.0;
                for (int kk = 0; kk < K; kk++)
                    c_cpu[ii * N + jj] += a[ii * K + kk] * b[kk * N + jj];
            }
        end_timer("CPU multiplication");

        bool flag = true;
        int mismatches = 0;
        for (int i = 0; i < M; i++)
        {
            for (int j = 0; j < N; j++){
                float x = c[i * N + j], y = c_cpu[i * N + j];
                if (x != y && ABS((x - y)) > EPSILON) // data_type precision comparision upto 10^-6 for types like doubles
                {
                    printf("(%d,%d) : got %lf expected %lf diff %e\n",i,j,x,y,ABS((x - y)));
                    flag = false;
                    mismatches++;
                    // break;
                }
            }
            // if (!flag)
            //     break;
        }
        printf("Correctness check: %s (mismatches = %d)\n",(flag ? "PASSED" : "FAILED"), mismatches);
        hipHostFree(c_cpu);
    }
    
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);

    for(auto &dev: streams)
        for(auto &str: dev)
            hipStreamDestroy(str);

    printf("DONE\n\n");
}