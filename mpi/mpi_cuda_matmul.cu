#include "hip/hip_runtime.h"
#include <mpi.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <vector>
#include <iostream>
#include <string>
#include <algorithm>
#include <atomic>
#include <mutex>
#include <thread>
#include <omp.h>
#include <cstdlib>
#include <numeric>
#include <cmath>
#include <ctime>

#define CEIL(x, y) (((x) + (y) - 1) / (y))
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define ABS(x) (((x) < (0)) ? (-x) : (x))
#define MAX_TPB 32

#define EPSILON 1e-4

#define PSIZE 2000

// Kernel for matrix-matrix multiplication
__global__ void multiply_kernel(float *A, float *B, float *C, int M, int N, int K)
{
    for(int i = blockIdx.y * blockDim.y + threadIdx.y;i<M;i+=blockDim.y*gridDim.y){
        for(int j = blockIdx.x * blockDim.x + threadIdx.x;j<N;j+=blockDim.x*gridDim.x)
    // if(i < M and j < N)
            {
                // atomicAdd(&d_counter, 1);
                float sum = 0.0;
                
                #if defined(VECTORIZE)
                auto a = reinterpret_cast<float4*>(&A[i * K]);
                auto b = reinterpret_cast<float4*>(&B[j * K]);
                // printf("check %d %d (%p %p) (%p %p) %d\n",i*K*4,j*K*4,a,&A[i * K],b,&B[j * K],K/4);
                for (int k = 0; k < K/4; k++)
                {
                    // printf("before\n");
                    // auto a = a_4[k], b = b_4[k];
                    // printf("%f,%f,%f,%f %f,%f,%f,%f\n",a.w,a.x,a.y,a.z,b.w,b.x,b.y,b.z);
                    sum += a->x*b->x + a->y*b->y + a->z*b->z + a->w*b->w;
                    // printf("(%f,%f)\n",a->w,b->w);
                    a++;
                    b++;
                }
                
                #else
                for (int k = 0; k < K; ++k)
                    sum += A[i * K + k] * B[j * K + k];
                #endif
                C[i * N + j] = sum;

                
            // printf("\n-------------\n");
            }
    }
}

__host__ inline hipError_t checkCuda(hipError_t status)
{
    if (status != hipSuccess)
    {
        std::cout << "CUDA Runtime Error: " << hipGetErrorString(status)
                  << std::endl;
        std::exit(EXIT_FAILURE);
    }
    return status;
}


void printMatrix(float *mat, int m, int n){
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%0.2lf ",mat[i*n+j]);
        }
        printf("\n");
    }
    printf("\n");
}

auto clk = std::chrono::high_resolution_clock::now();

void start_timer(){
    clk = std::chrono::high_resolution_clock::now();
}

void end_timer(std::string func){
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - clk);
    std::cout<<func<<" took "<<1.0e-9 * duration.count()<<" seconds\n";
}

std::vector<int> generateEqualChunkStartIndices(int n, int m) {
    std::vector<int> startIndexes;
    int baseSize = n / m;               // Base size of each chunk
    int remainder = n % m;              // Remainder to be distributed
    int startIndex = 0;

    // Generate starting indices based on uniform chunk sizes
    for (int i = 0; i < m; ++i) {
        startIndexes.push_back(startIndex);
        int currentChunkSize = baseSize + (i < remainder ? 1 : 0);  // Distribute remainder among the first few chunks
        startIndex += currentChunkSize;
    }

    return startIndexes;
}

// Function to calculate chunk sizes from start indices
std::vector<int> calculateChunkSizes(const std::vector<int>& startIndexes, int n) {
    std::vector<int> chunkSizes;
    for (size_t i = 0; i < startIndexes.size(); ++i) {
        if (i == startIndexes.size() - 1) {
            chunkSizes.push_back(n - startIndexes[i]);  // Last chunk goes to the end of the array
        } else {
            chunkSizes.push_back(startIndexes[i + 1] - startIndexes[i]);
        }
    }
    return chunkSizes;
}

void checkMPIError(int status) {
    if (status != MPI_SUCCESS) {
        char errorString[MPI_MAX_ERROR_STRING];
        int lengthOfErrorString;
        MPI_Error_string(status, errorString, &lengthOfErrorString);

        std::cerr << "Error code: " << status << "\n"
                  << "Error description: " << errorString << std::endl;

        MPI_Abort(MPI_COMM_WORLD, status);  // Abort MPI execution
        std::exit(EXIT_FAILURE);            // Terminate the program
    }
}


void transposeMatrix(float* matrix, int m, int n) {
    for (int i = 0; i < m; ++i) {
        for (int j = i + 1; j < n; ++j) {
            std::swap(matrix[i * n + j], matrix[j * n + i]);
        }
    }
}

int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);

    int ndevs = 0;
    hipError_t error_id = hipGetDeviceCount(&ndevs);

    int world_size, world_rank;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
    int check_result = 0;
    // printf("world size %d rank %d GPUs %d\n",world_size,world_rank,ndevs);

    int M = PSIZE, N = PSIZE, K = PSIZE;
    if (argc <= 1)
    {
        printf("Usage bench_works [m] [n] [k]\n");
        printf("Using default parameters\n");
    }
    else
    {
        if (argc > 1)
            M = atoi(argv[1]);
        if (argc > 2)
            N = atoi(argv[2]);
        if (argc > 3)
            K = atoi(argv[3]);
        if (argc > 4)
            check_result = atoi(argv[4]);
    }
    int a_size = M * K, b_size = K * N, c_size = M * N;

    int numRowsPerRank = CEIL(M,world_size);
    

    
    std::vector<int> startIndexes = generateEqualChunkStartIndices(M, world_size);;
    std::vector<int> chunkSizes = calculateChunkSizes(startIndexes, M);

    hipSetDevice(world_rank%ndevs);
    checkMPIError(MPI_Barrier(MPI_COMM_WORLD));
        
    int start = startIndexes[world_rank], end = (world_rank==world_size-1 ? M : startIndexes[world_rank+1]);
    int nRows = end-start;
    int a_start, b_start, c_start, a_items, b_items, c_items, m, n, k;
    
    m=nRows; n=N; k=K;
    a_start = start*K; b_start = 0;   c_start = start*N;
    a_items = nRows*K; b_items = K*N; c_items = nRows*N;
    MPI_Status stat;

    float *a,*b,*c;
    if (world_rank == 0) {

        checkCuda(hipHostMalloc(&a,a_size*sizeof(float)));
        checkCuda(hipHostMalloc(&b,b_size*sizeof(float)));
        checkCuda(hipHostMalloc(&c,c_size*sizeof(float)));

        // initialize

        for (int i = 0; i < a_size; i++)
            // a[i] = (float)rand() / RAND_MAX * 2.0 - 1.0;
            a[i] = i%7;

        for (int i = 0; i < b_size; i++)
            // b[i] = (float)rand() / RAND_MAX * 2.0 - 1.0;
            b[i] = 9+i%8;

        for (int i = 0; i < c_size; i++)
            c[i] = 0.0;

        start_timer();
        
        printf("bench_works [m=%d] [n=%d] [k=%d]\n",M, N, K);
        #if defined(VECTORIZE)
        printf("vectorized\n");   
        #endif
    }


    float *h_a, *h_b, *h_c;
    if(world_rank==0){
        h_a = a+a_start;
        h_b = b+b_start;
        h_c = c+c_start;
        for(int i=1;i<world_size;i++){
            int send_start = startIndexes[i], send_end = (i==world_size-1 ? M : startIndexes[i+1]);
            int send_nRows = send_end-send_start;
            checkMPIError(MPI_Send(a+send_start*K   ,send_nRows*K   ,MPI_FLOAT,i,1,MPI_COMM_WORLD));
            checkMPIError(MPI_Send(b+0              ,K*N            ,MPI_FLOAT,i,2,MPI_COMM_WORLD));
            checkMPIError(MPI_Send(c+send_start*N   ,send_nRows*N   ,MPI_FLOAT,i,3,MPI_COMM_WORLD));
        }
        
    }
    else{
        
        checkCuda(hipHostMalloc(&h_a,a_items*sizeof(float)));
        checkCuda(hipHostMalloc(&h_b,b_items*sizeof(float)));
        checkCuda(hipHostMalloc(&h_c,c_items*sizeof(float)));

        checkMPIError(MPI_Recv(h_a,a_items,MPI_FLOAT,0,1,MPI_COMM_WORLD,&stat));
        checkMPIError(MPI_Recv(h_b,b_items,MPI_FLOAT,0,2,MPI_COMM_WORLD,&stat));
        checkMPIError(MPI_Recv(h_c,c_items,MPI_FLOAT,0,3,MPI_COMM_WORLD,&stat));
    }
    
    // checkMPIError(MPI_Barrier(MPI_COMM_WORLD));
    // int p = 2;
    // if(world_rank==p){
    //     printMatrix(h_a,m,k);
    // }


    transposeMatrix(h_b,K,N);

    float *d_a,*d_b,*d_c;

    hipMalloc(&d_a,a_items*sizeof(float));
    hipMemcpy(d_a,h_a,a_items*sizeof(float),hipMemcpyHostToDevice);
    
    hipMalloc(&d_b,b_items*sizeof(float));
    hipMemcpy(d_b,h_b,b_items*sizeof(float),hipMemcpyHostToDevice);
    
    hipMalloc(&d_c,c_items*sizeof(float));
    hipMemcpy(d_c,h_c,c_items*sizeof(float),hipMemcpyHostToDevice);

    int blk_x = min(MAX_TPB,n), blk_y = min(MAX_TPB,m);
    dim3 blocksPerGrid(CEIL(n,blk_x), CEIL(m,blk_y));
    dim3 threadsPerBlock(blk_x, blk_y); // Assuming width and height are within max threads per block limit 

    multiply_kernel<<<blocksPerGrid,threadsPerBlock>>>(d_a,d_b,d_c,m,n,k);

    hipMemcpy(h_c,d_c,c_items*sizeof(float),hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipDeviceSynchronize();

    transposeMatrix(h_b,N,K);
    
    // int rank = 0;
    // while (rank < world_size) {
    //     if (world_rank == rank) {
    //         printf("rank %d startrow %d endrow %d a_start %d b_start %d c_start %d\n",world_rank,start,end,a_start,b_start,c_start);
    //         printf ("Array printed by rank: %d\n", world_rank);
    //         printMatrix(h_c,m,n);
    //         fflush(stdout);
    //     }
    //     rank++;
    //     MPI_Barrier(MPI_COMM_WORLD);
    // }

    if(world_rank==0){
        h_a = a+a_start;
        h_b = b+b_start;
        h_c = c+c_start;
        for(int i=1;i<world_size;i++){
            int recv_start = startIndexes[i], recv_end = (i==world_size-1 ? M : startIndexes[i+1]);
            int recv_nRows = recv_end-recv_start;
            checkMPIError(MPI_Recv(c+recv_start*N, recv_nRows*N,MPI_FLOAT,i,4,MPI_COMM_WORLD,&stat));
        }
    }
    else{
        checkMPIError(MPI_Send(h_c,c_items,MPI_FLOAT,0,4,MPI_COMM_WORLD));
    }

    if(world_rank==0){
        end_timer("GPU Multiplication");
        if(check_result){
            float *c_cpu;
            // checkCuda(hipHostMalloc(&c_cpu,c_size*sizeof(float)));
            c_cpu = (float*)malloc(c_size*sizeof(float));
            start_timer();
            printf("GPU Done... now checking correctness\n");
            #pragma omp parallel for
            for (int ii = 0; ii < M; ii++)
                for (int jj = 0; jj < N; jj++){
                    c_cpu[ii * N + jj] = 0.0;
                    for (int kk = 0; kk < K; kk++)
                        c_cpu[ii * N + jj] += a[ii * K + kk] * b[kk * N + jj];
                }
            end_timer("CPU multiplication");

            // printMatrix(c_cpu,M,N);

            bool flag = true;
            int mismatches = 0;
            for (int i = 0; i < M; i++)
            {
                for (int j = 0; j < N; j++){
                    float x = c[i * N + j], y = c_cpu[i * N + j];
                    if (x != y && ABS((x - y)) > EPSILON) // data_type precision comparision upto 10^-6 for types like doubles
                    {
                        printf("(%d,%d) : got %lf expected %lf diff %e\n",i,j,x,y,ABS((x - y)));
                        flag = false;
                        mismatches++;
                        // break;
                    }
                }
                // if (!flag)
                //     break;
            }
            printf("Correctness check: %s (mismatches = %d)\n",(flag ? "PASSED" : "FAILED"), mismatches);
            hipHostFree(c_cpu);
        }
        hipHostFree(a);
        hipHostFree(b);
        hipHostFree(c);
    }
    else{
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
    }

    MPI_Finalize();
    return 0;
}
